#include "hip/hip_runtime.h"
#include <dirent.h>
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <fstream>
#include <chrono>
#include <float.h>
#include <limits.h>
#include <math.h>
#include "nvtx.cuh"

#include "api.hh"

#include "cli/quality_viewer.hh"
#include "cli/timerecord_viewer.hh"

using Compressor = typename cusz::Framework<float>::LorenzoFeaturedCompressor;

//#define MIN(x, y) ((x) < (y) ? (x) : (y))
#define MAX(x, y) ((x) > (y) ? (x) : (y))

typedef struct
{
    float *h_uncompressed_data;
    float *d_uncompressed_data;
    float *h_decompressed_data;
    float *d_decompressed_data;
    uint8_t *h_compressed_data;
    uint8_t *d_compressed_data;
    double eb;
    int device;
    size_t uncompressed_len;
    size_t compressed_len;
    cusz::Header header;
    char *mode;
} Data_t;

void compress(Data_t *data, size_t nx, size_t ny, size_t nz, hipStream_t stream)
{
    size_t uncompressed_alloclen = data->uncompressed_len * 1.03;

    // Defining cusz stuff
    Compressor *compressor = new Compressor;
    cusz::TimeRecord timerecord;
    cusz::Context *ctx = new cusz::Context();
    ctx->set_len(nx, ny, nz, 1).set_eb(data->eb).set_control_string(data->mode);
    ctx->device = data->device;

    float *d_uncompressed_copy;
    hipMalloc(&d_uncompressed_copy, sizeof(float) * nx * ny * nz);
    hipMemcpy(d_uncompressed_copy, data->d_uncompressed_data, sizeof(float) * nx * ny * nz, hipMemcpyHostToDevice);

    cusz::Context::adjust_eb(ctx, d_uncompressed_copy);

    NVTX_PUSH_RANGE("CUSZ_COMPRESS", MY_YELLOW);
    cusz::core_compress(compressor, ctx,                                             // compressor & config
                        d_uncompressed_copy, uncompressed_alloclen,                  // input
                        data->d_compressed_data, data->compressed_len, data->header, // output
                        stream, &timerecord);
    NVTX_POP_RANGE();

    hipFree(d_uncompressed_copy);
    delete compressor;
}

void decompress(Data_t *data, hipStream_t stream)
{
    auto compressor = new Compressor;
    cusz::TimeRecord timerecord;
    size_t uncompressed_alloclen = data->uncompressed_len * 1.03;

    hipMalloc(&data->d_uncompressed_data, sizeof(float) * uncompressed_alloclen);

    NVTX_PUSH_RANGE("CUSZ_DECOMPRESS", MY_YELLOW);
    cusz::core_decompress(compressor, &data->header,
                          data->d_compressed_data,   // input
                          data->compressed_len,      // input len
                          data->d_decompressed_data, // output
                          uncompressed_alloclen,     // output len
                          stream, &timerecord);
    NVTX_POP_RANGE();

    // cusz::TimeRecordViewer::view_decompression(&timerecord, len * sizeof(float));

    delete compressor;
}

void readInputDataFromFile(string filepath, float *h_array, size_t len)
{
    std::ifstream ifs(filepath.c_str(), std::ios::binary | std::ios::in);
    if (not ifs.is_open())
    {
        std::cerr << "fail to open " << filepath << std::endl;
        exit(1);
    }
    ifs.read(reinterpret_cast<char *>(h_array), std::streamsize(len * sizeof(float)));
    ifs.close();
}

void exportData(string path, void *h_data, int data_size, size_t len)
{
    auto file = fopen(path.c_str(), "wb");
    fwrite(h_data, data_size, len, file);
    fclose(file);
}

int main(int argc, char *argv[])
{
    int gpus = 4;
    int iterationsPerGpu = 10;
    double eb = 1e-4;
    char *mode = "mode=abs"; // "abs" or "r2r"
    string inputFilepath = "../../hurr-CLOUDf48-500x500x100";
    size_t nx = 500;
    size_t ny = 500;
    size_t nz = 100;
    bool dumpData = false;
    bool printReport = false;

    fprintf(stderr, "----------CUSZ------------------\n");
    fprintf(stderr, "Parameters\n");
    fprintf(stderr, "EB: %lf; Mode: %s\n", eb, mode);
    fprintf(stderr, "# GPUs: %i; # iterations per GPU: %i\n", gpus, iterationsPerGpu);
    fprintf(stderr, "Input file path %s\n", inputFilepath.c_str());
    fprintf(stderr, "Dims: (%li, %li, %li)\n", nx, ny, nz);
    fprintf(stderr, "--------------------------------\n");

    int gpu = 0;
    for (int i = 0; i < gpus * iterationsPerGpu; i++)
    {
        fprintf(stderr, "Iteration #%i; GPU #%i\n", i, gpu);

        hipSetDevice(gpu);

        Data_t _data;
        Data_t *data = &_data;
        hipStream_t stream;
        hipStreamCreate(&stream);

        size_t len = nx * ny * nz;
        data->uncompressed_len = len;
        data->eb = eb;
        data->mode = mode;
        data->device = gpu;

        hipHostMalloc(&data->h_uncompressed_data, len * sizeof(float));
        readInputDataFromFile(inputFilepath, data->h_uncompressed_data, len);

        hipMalloc(&data->d_uncompressed_data, len * sizeof(float));
        hipMemcpy(data->d_uncompressed_data, data->h_uncompressed_data, len * sizeof(float), hipMemcpyHostToDevice);

        chrono::steady_clock::time_point begin;
        chrono::steady_clock::time_point end;

        begin = std::chrono::steady_clock::now();
        NVTX_PUSH_RANGE("START_COMPRESSION_METHOD", MY_ORANGE);
        compress(data, nx, ny, nz, stream);
        NVTX_POP_RANGE();
        end = std::chrono::steady_clock::now();
        fprintf(stderr, "Compression spent time %li[µs]\n", chrono::duration_cast<chrono::microseconds>(end - begin).count());

        fprintf(stderr, "Starting decompression\n");
        begin = std::chrono::steady_clock::now();
        NVTX_PUSH_RANGE("START_DECOMPRESSION_METHOD", MY_ORANGE);
        hipMalloc(&data->d_decompressed_data, len * sizeof(float));
        decompress(data, stream);
        NVTX_POP_RANGE();
        end = std::chrono::steady_clock::now();
        fprintf(stderr, "DEcompression spent time %li[µs]\n", chrono::duration_cast<chrono::microseconds>(end - begin).count());

        if (dumpData)
        {
            exportData("./dump/decompressed-from-api_" + std::to_string(i), data->h_decompressed_data, sizeof(float), len);
            hipHostMalloc(&data->h_compressed_data, data->compressed_len);
            hipMemcpy(data->h_compressed_data, data->d_compressed_data, data->compressed_len, hipMemcpyDeviceToHost);
            exportData("./dump/compressed-from-api_" + std::to_string(i), data->h_compressed_data, 1, data->compressed_len);
        }

        if (printReport)
        {
            fprintf(stderr, "Report:\n");
            hipHostFree(&data->h_decompressed_data);
            hipHostMalloc(&data->h_decompressed_data, len * sizeof(float));
            hipMemcpy(data->h_decompressed_data, data->d_decompressed_data, len * sizeof(float), hipMemcpyHostToDevice);

            fprintf(stderr, "CPU Metrics:\n");
            cusz::QualityViewer::echo_metric_cpu(data->h_decompressed_data, data->h_uncompressed_data, len, size_t(data->compressed_len), false);
        }

        hipHostFree(&data->h_decompressed_data);
        hipFree(data->d_decompressed_data);
        hipHostFree(&data->h_uncompressed_data);
        hipFree(data->d_uncompressed_data);
        hipHostFree(&data->h_compressed_data);
        hipFree(data->d_compressed_data);
        hipStreamDestroy(stream);

        if (gpus > 1)
        {
            if (gpu + 1 < gpus)
            {
                gpu++;
            }
            else
            {
                gpu = 0;
            }
        }

        fprintf(stderr, "\n----------------------------------------\n\n");
    }
}
